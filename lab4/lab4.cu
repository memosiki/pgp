
#include <hip/hip_runtime.h>
#include <numeric>
#include <stdio.h>
#include <math.h>
#include <algorithm>
#include <string.h>

#define MAX_N 1000

#define CSC(call)                                                    \
do {                                                                \
    hipError_t res = call;                                            \
    if (res != hipSuccess) {                                        \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n",            \
                __FILE__, __LINE__, hipGetErrorString(res));        \
        exit(0);                                                    \
    }                                                                \
} while(0)
__constant__ double cur_row[MAX_N];

__global__ void kernel(double* dev_a, int i, int n){
    int start_x = gridDim.x * blockIdx.x + threadIdx.x;
    int start_y = gridDim.y * blockIdx.y + threadIdx.y;
    int offset_x = gridDim.x * blockDim.x;
    int offset_y = gridDim.y * blockDim.y;

    double ratio;
    for (int y = i + 1 + start_y; y < n; y += offset_y) {
        ratio = dev_a[y * MAX_N + i] / cur_row[i];
        for (int x = i + 1 + start_x; x < n; x += offset_x){
            dev_a[y * MAX_N + x] -= ratio * cur_row[x];
            // printf("upd %d %d\n",x, y );
        }
    }
}
double a[MAX_N][MAX_N];
int cur_i;
bool cmp(int el1, int el2){
    // компаратор -- сравнивает значения в столбцах у строк по заданным значениям
    // номер столбца задаётся глобально
    return abs(a[el1][cur_i]) < abs(a[el2][cur_i]);
}
int main() {
     // double a[10000][10000];
    int n;
    double eps = 1e-7;
    scanf("%d", &n);
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < n; ++j)
            scanf("%lf", &a[i][j]);
            // храним по строкам
    double *dev_a;
    CSC(hipMalloc( (void**) &dev_a, sizeof(double) * MAX_N * MAX_N));
    int pos[MAX_N];
    std::iota(pos, pos+n, 0);
    int sign = 1;
    double det = 0;
    for (int i = 0; i < n; ++i) {
        // выбираем максимальный
        cur_i = i;
        int* it;
        it = std::max_element(pos+i, pos+n, cmp);
        int major = it - pos;


        // printf("%d %d %lf\n", i, major, a[major][i]);
        if (i == n-1){
            // если последняя строка
            if (abs(a[i][i]) < eps ){
                printf("det %.10e\n", 0.);
                return 0;
            }
            else if (a[i][i] < 0){
                det += log( -a[i][i] );
                sign = - sign;
            } else
                det += log( a[i][i] );
            break;
        }
        CSC(hipMemcpyToSymbol(HIP_SYMBOL(cur_row), a[major], sizeof(double) * n, 0, hipMemcpyHostToDevice));

        if (abs(a[major][i]) < eps ){
            printf("det %.10e\n", 0.);
            return 0;
        } else if (a[major][i] < 0){
            det += log( -a[major][i] );
            sign = - sign;
        } else
            det += log( a[major][i] );

        if (major != i){
            memcpy((double*)a + major * MAX_N,  (double*)a + i * MAX_N, MAX_N * sizeof(double));
            sign = -sign;
        }
        // CSC(cudaMemcpy(dev_a + i * MAX_N, a + i * MAX_N, sizeof(double) * MAX_N * (MAX_N - i), cudaMemcpyHostToDevice));
        CSC(hipMemcpy(dev_a, a, sizeof(double) * MAX_N * MAX_N, hipMemcpyHostToDevice));
        // printf("aaaaa\n");

        //вычитаем из каждой строчки текущую
        kernel <<< dim3(32, 32), dim3(32, 32) >>> (dev_a, i, n);

        CSC(hipGetLastError());
        // CSC(cudaMemcpy((double*)a + i * MAX_N, (double*) dev_a + i * MAX_N, sizeof(double) * MAX_N * (MAX_N - i), cudaMemcpyDeviceToHost));
        CSC(hipMemcpy(a, dev_a, sizeof(double) * MAX_N * MAX_N, hipMemcpyDeviceToHost));

    }
    double ans = sign*exp(det);
    printf("det %.10e\n", ans);
    return 0;
}
