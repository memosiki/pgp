#include <stdio.h>
#include <string.h>
#include "image.h"
#include "classifier.h"
#include "kernel.h"
#include "csc.h"

int classify() {
    char inputFile[256];
    char outputFile[256];
    scanf("%s", inputFile);
    scanf("%s", outputFile);
    Image *inputImage = readImageFromFile(inputFile);
    Pixel* outputPixels;
    int size = sizeof(Pixel) * inputImage->width * inputImage->height;
    CSC(hipMalloc(&outputPixels, size));
    CSC(hipMemcpy(outputPixels, inputImage->pixels, size, hipMemcpyHostToDevice));
    Classifier *c = createClassifier(inputImage);
    copyClassifierToConstant(c);
    dim3 gridSize(16, 16);
	dim3 blockSize(16, 16);
	classifyPixels<<<gridSize, blockSize>>>(outputPixels, inputImage->width, inputImage->height);
    CSC(hipGetLastError());
    CSC(hipMemcpy(inputImage->pixels, outputPixels, size, hipMemcpyDeviceToHost));
    writeImageToFile(inputImage, outputFile);
    deleteClassifier(c);
    deleteImage(inputImage);
    return 0;
}

int main(void)
{
    classify();
	return 0;
}